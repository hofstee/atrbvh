#include "CubWrapper.h"

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>
#include "Defines.h"

namespace BVHRT
{

// In and out buffers may be swaped
// Original data is not kept
template <typename T> float DeviceSort(unsigned int numberOfElements, T** keysIn, T** keysOut,
                 unsigned int** valuesIn, unsigned int** valuesOut)
{
    //hipcub::DoubleBuffer<T> keysBuffer(*keysIn, *keysOut);
    //hipcub::DoubleBuffer<unsigned int> valuesBuffer(*valuesIn, *valuesOut);

    // Check how much temporary memory will be required
    void* tempStorage = nullptr;
    size_t storageSize = 0;
    // hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
    // numberOfElements);
    //hipcub::DeviceRadixSort::SortKeys(tempStorage, storageSize, keysBuffer, numberOfElements);
    hipcub::DeviceRadixSort::SortKeys(tempStorage, storageSize, *keysIn, *keysOut, numberOfElements);

    // Allocate temporary memory
    hipMalloc(&tempStorage, storageSize);

    float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
#endif

    // Sort
    //hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
    //    numberOfElements);
    hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, *keysIn, *keysOut, *valuesIn, *valuesOut,
        numberOfElements);

#ifdef MEASURE_EXECUTION_TIMES
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
#endif

    // Free temporary memory
    hipFree(tempStorage);

    //// Update out buffers
    //T* current = keysBuffer.Current();
    //keysOut = &current;
    //unsigned int* current2 = valuesBuffer.Current();
    //valuesOut = &current2;

    //// Update in buffers
    //current = keysBuffer.d_buffers[1 - keysBuffer.selector];
    //keysIn = &current;
    //current2 = valuesBuffer.d_buffers[1 - valuesBuffer.selector];
    //valuesIn = &current2;

    return elapsedTime;
}

float DeviceSort(unsigned int numberOfElements, unsigned int** keysIn, unsigned int** keysOut,
    unsigned int** valuesIn, unsigned int** valuesOut)
{
    return DeviceSort<unsigned int>(numberOfElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSort(unsigned int numberOfElements, unsigned long long int** keysIn, unsigned long long int** keysOut,
    unsigned int** valuesIn, unsigned int** valuesOut)
{
    return DeviceSort<unsigned long long int>(numberOfElements, keysIn, keysOut, valuesIn, valuesOut);
}

void DeviceSum(unsigned int numberOfTriangles, int* in, int* out, size_t* tempMemorySize, void* tempMemory)
{
    hipcub::DeviceReduce::Sum(tempMemory, *tempMemorySize, in, out, numberOfTriangles);
}

template <class T> T DeviceSum(unsigned int numberOfElements, T* elements)
{
    T* deviceElementsSum;
    hipMalloc(&deviceElementsSum, sizeof(T));

    // Calculate the required temporary memory size
    void* tempStorage = nullptr;
    size_t tempStorageSize = 0;
    hipcub::DeviceReduce::Sum(tempStorage, tempStorageSize, elements, deviceElementsSum,
                           numberOfElements);

    // Allocate temporary memory
    hipMalloc(&tempStorage, tempStorageSize);

    // Sum priorities
    hipcub::DeviceReduce::Sum(tempStorage, tempStorageSize, elements, deviceElementsSum,
                           numberOfElements);

    // Read priorities sum from device memory
    T elementsSum;
    hipMemcpy(&elementsSum, deviceElementsSum, sizeof(T), hipMemcpyDeviceToHost);

    // Free temporary memory
    hipFree(tempStorage);
    hipFree(deviceElementsSum);

    return elementsSum;
}

int DeviceSum(unsigned int numberOfElements, int* elements)
{
    return DeviceSum<int>(numberOfElements, elements);
}

float DeviceSum(unsigned int numberOfElements, float* elements)
{
    return DeviceSum<float>(numberOfElements, elements);
}
}
